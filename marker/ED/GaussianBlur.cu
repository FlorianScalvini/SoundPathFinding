
#include <hip/hip_runtime.h>
//
// Created by Florian SCALVINI on 26/06/22.
//


__global__ void convolution(char * img, float * img_out, const float * __restrict__ mask,
                            const int halfKernel, const int width, const int height)
{
    extern __shared__ float lInput[];


    const unsigned int indexRowG = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int indexColG = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int localRowLen = 2*halfKernel + blockDim.x;
    const unsigned int idxLocal = halfKernel + threadIdx.x + localRowLen * (threadIdx.y + halfKernel);
    const unsigned int idx = indexColG + width * indexRowG ;

    if(indexColG >= width && indexRowG >= height)
    {
        lInput[idxLocal] = 0;
    }
    else
    {
        lInput[idxLocal] = img[idx];
    }



    __syncthreads();
    int fIndex = 0;
    float sum = 0;
    for (int row = -halfKernel; row <= halfKernel; row+=1)
    {
        int curRow = row * localRowLen;
        for (int col = -halfKernel; col <= halfKernel; col+=1, fIndex++)
        {
            sum += lInput[idxLocal + curRow + col] * mask[fIndex];
        }
    }

    img_out[idx] = sum;
    return;
}